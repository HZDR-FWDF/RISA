#include "hip/hip_runtime.h"
/*
 * This file is part of the RISA-library.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * RISA is free software: You can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * RISA is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with RISA. If not, see <http://www.gnu.org/licenses/>.
 *
 * Date: 30 November 2016
 * Authors: Tobias Frust (FWCC) <t.frust@hzdr.de>
 *
 */

#include <risa/Reordering/Reordering.h>
#include <risa/ConfigReader/ConfigReader.h>
#include <risa/Basics/performance.h>

#include <glados/cuda/Launch.h>
#include <glados/cuda/Check.h>
#include <glados/cuda/Coordinates.h>
#include <glados/MemoryPool.h>

#include <boost/log/trivial.hpp>

#include <nvToolsExt.h>

#include <exception>
#include <pthread.h>

namespace risa {
namespace cuda {

__global__ void reorder(const unsigned short* __restrict__ unorderedSino, unsigned short* __restrict__ orderedSino,
      const int* __restrict__ hashTable, const int numberOfProjections, const int numberOfDetectors);

Reordering::Reordering(const std::string& configFile) {

   if (readConfig(configFile)) {
      throw std::runtime_error(
            "recoLib::cuda::CropImage: Configuration file could not be loaded successfully. Please check!");
   }

   CHECK(hipGetDeviceCount(&numberOfDevices_));

   //custom streams are necessary, because profiling with nvprof not possible with
   //-default-stream per-thread option
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      //register in memory pool
      memoryPoolIdxs_[i] = glados::MemoryPool<deviceManagerType>::instance()->registerStage(memPoolSize_, numberOfFanDetectors_*numberOfFanProjections_);
      hipStream_t stream;
      CHECK(hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, 6));
      streams_[i] = stream;
   }

   //initialize worker threads
   for (auto i = 0; i < numberOfDevices_; i++) {
      processorThreads_[i] = std::thread { &Reordering::processor, this, i };
   }
   BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::CropImage: Running " << numberOfDevices_ << " Threads.";
}

Reordering::~Reordering() {
   for (auto idx : memoryPoolIdxs_) {
      CHECK(hipSetDevice(idx.first));
      glados::MemoryPool<deviceManagerType>::instance()->freeMemory(idx.second);
   }
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      CHECK(hipStreamDestroy(streams_[i]));
   }
   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::CropImage: Destroyed.";
}

auto Reordering::process(input_type&& img) -> void {
   if (img.valid()) {
      BOOST_LOG_TRIVIAL(debug)<< "CropImage: Image arrived with Index: " << img.index() << "to device " << img.device();
      sinos_[img.device()].push(std::move(img));
   } else {
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::CropImage: Received sentinel, finishing.";

      //send sentinal to processor thread and wait 'til it's finished
      for(auto i = 0; i < numberOfDevices_; i++) {
         sinos_[i].push(input_type());
      }

      for(auto i = 0; i < numberOfDevices_; i++) {
         processorThreads_[i].join();
      }
      //push sentinel to results for next stage
      results_.push(output_type());
      BOOST_LOG_TRIVIAL(info) << "recoLib::cuda::CropImage: Finished.";
   }
}

auto Reordering::wait() -> output_type {
   return results_.take();
}

/**
 * The processor()-Method takes one sinogram from the queue. Via the cuFFT-Library
 * it is transformed into frequency space for applying the filter function.
 * After filtering the transformation is reverted via the inverse fourier transform.
 * Finally, the filtered sinogram is pushed back into the output queue for
 * further processing.
 *
 */
auto Reordering::processor(const int deviceID) -> void {
   //nvtxNameOsThreadA(pthread_self(), "CropImage");
   CHECK(hipSetDevice(deviceID));
   dim3 blocks(16, 16);
   dim3 grids(std::ceil(numberOfFanDetectors_/16.0),
         std::ceil(numberOfFanProjections_/16.0));

   std::vector<int> hashTable(numberOfFanDetectors_*numberOfFanProjections_);
   createHashTable(hashTable);

   auto d_hashTable = glados::cuda::make_device_ptr<int>(numberOfFanDetectors_*numberOfFanProjections_);
   CHECK(hipMemcpy(d_hashTable.get(), hashTable.data(), sizeof(int)*hashTable.size(), hipMemcpyHostToDevice));

   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::Reordering: Running Thread for Device " << deviceID;
   while (true) {
      auto img = sinos_[deviceID].take();
      if (!img.valid())
         break;
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Reordering: Reordering image with Index " << img.index();

      auto sino_ordered = glados::MemoryPool<deviceManagerType>::instance()->requestMemory(memoryPoolIdxs_[deviceID]);

      reorder<<<grids, blocks, 0, streams_[deviceID]>>>(img.container().get(), sino_ordered.container().get(), d_hashTable.get(), numberOfFanProjections_, numberOfFanDetectors_);
      CHECK(hipPeekAtLastError());

      sino_ordered.setIdx(img.index());
      sino_ordered.setDevice(img.device());
      sino_ordered.setPlane(img.plane());
      sino_ordered.setStart(img.start());

      //wait until work on device is finished
      CHECK(hipStreamSynchronize(streams_[deviceID]));
      results_.push(std::move(sino_ordered));

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Reordering: Reordering image with Index " << img.index() << " finished.";
   }
}

/**
 * All values needed for setting up the class are read from the config file
 * in this function.
 *
 * @param[in] configFile path to config file
 *
 * @return returns true, if configuration file could be read successfully, else false
 */
auto Reordering::readConfig(const std::string& configFile) -> bool {
   ConfigReader configReader = ConfigReader(configFile.data());
   int samplingRate, scanRate;
   if (configReader.lookupValue("numberOfFanDetectors", numberOfFanDetectors_)
         && configReader.lookupValue("memPoolSize_Reordering", memPoolSize_)
         && configReader.lookupValue("samplingRate", samplingRate)
         && configReader.lookupValue("scanRate", scanRate)){
      numberOfDetectorsPerModule_ = 16;
      numberOfFanProjections_ = samplingRate * 1000000 / scanRate;
      return EXIT_SUCCESS;
   }
   else
      return EXIT_FAILURE;

}

auto Reordering::createHashTable(std::vector<int>& hashTable) -> void {
   int numberOfModules = 27;
   int i = 0;
   hashTable.resize(numberOfFanProjections_*numberOfFanDetectors_);
   for(auto projInd = 0; projInd < numberOfFanProjections_; projInd++){
      for(auto modInd = 0; modInd < numberOfModules; modInd++){
         for(auto detInd = 0; detInd < numberOfDetectorsPerModule_; detInd++){
            int index = detInd + projInd * numberOfDetectorsPerModule_ + modInd * numberOfDetectorsPerModule_*numberOfFanProjections_;
            hashTable[i] = index;
            i++;
         }
      }
   }
}

__global__ void reorder(const unsigned short* __restrict__ unorderedSino, unsigned short* __restrict__ orderedSino,
      const int* __restrict__ hashTable, const int numberOfProjections, const int numberOfDetectors) {
   const auto x = glados::cuda::getX();
   const auto y = glados::cuda::getY();
   if (x >= numberOfDetectors || y >= numberOfProjections)
      return;

   const int index = x + y * numberOfDetectors;
   orderedSino[index] = unorderedSino[hashTable[index]];
}

}
}

