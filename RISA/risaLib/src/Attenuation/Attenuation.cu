#include "hip/hip_runtime.h"
/*
 * This file is part of the RISA-library.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * RISA is free software: You can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * RISA is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with RISA. If not, see <http://www.gnu.org/licenses/>.
 *
 * Date: 30 November 2016
 * Authors: Tobias Frust <t.frust@hzdr.de>
 *
 */

#include "../DetectorInterpolation/interpolationFunctions.h"
#include "../../include/risa/Attenuation/Attenuation.h"
#include "../../include/risa/Basics/performance.h"

#include <glados/cuda/Launch.h>
#include <glados/cuda/Check.h>
#include <glados/cuda/Coordinates.h>
#include <glados/MemoryPool.h>

#include <boost/log/trivial.hpp>

#include <omp.h>
#include <iostream>
#include <cmath>
#include <fstream>
#include <iterator>
#include <exception>
#include <pthread.h>

namespace risa {
namespace cuda {

Attenuation::Attenuation(const std::string& config_file) {

   risa::read_json config_reader{};
   config_reader.read(config_file);

   if (readConfig(config_reader)) {
      throw std::runtime_error(
            "recoLib::cuda::Attenuation: Configuration file could not be loaded successfully. Please check!");
   }

   numberOfDarkFrames_ = 500;

   CHECK(hipGetDeviceCount(&numberOfDevices_));

   //custom streams are necessary, because profiling with nvprof not possible with
   //-default-stream per-thread option
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      memoryPoolIdxs_[i] =
            glados::MemoryPool<deviceManagerType>::instance()->registerStage(memPoolSize_,
                  numberOfDetectors_ * numberOfProjections_);
      hipStream_t stream;
      CHECK(hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, 5));
      streams_[i] = stream;
   }

   init();

   //initialize worker threads
   for (auto i = 0; i < numberOfDevices_; i++) {
      processorThreads_[i] = std::thread { &Attenuation::processor, this, i };
   }

   BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Attenuation: Running " << numberOfDevices_ << " Threads.";
}

Attenuation::~Attenuation() {
   for (auto idx : memoryPoolIdxs_) {
      CHECK(hipSetDevice(idx.first));
      glados::MemoryPool<deviceManagerType>::instance()->freeMemory(idx.second);
   }
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      CHECK(hipStreamDestroy(streams_[i]));
   }
   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::Attenuation: Destroyed.";
}

auto Attenuation::process(input_type&& sinogram) -> void {
   if (sinogram.valid()) {
      BOOST_LOG_TRIVIAL(debug)<< "Attenuation: Image arrived with Index: " << sinogram.index() << "to device " << sinogram.device();
      sinograms_[sinogram.device()].push(std::move(sinogram));
   } else {
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Attenuation: Received sentinel, finishing.";

      //send sentinal to processor thread and wait 'til it's finished
      for(auto i = 0; i < numberOfDevices_; i++) {
         sinograms_[i].push(input_type());
      }

      for(auto i = 0; i < numberOfDevices_; i++) {
         processorThreads_[i].join();
      }
      //push sentinel to results for next stage
      results_.push(output_type());
      BOOST_LOG_TRIVIAL(info) << "recoLib::cuda::Attenuation: Finished.";
   }
}

auto Attenuation::wait() -> output_type {
   return results_.take();
}

auto Attenuation::processor(const int deviceID) -> void {
   //nvtxNameOsThreadA(pthread_self(), "Attenuation");
   CHECK(hipSetDevice(deviceID));
   auto avgDark_d = glados::cuda::make_device_ptr<float>(avgDark_.size());
   auto avgReference_d = glados::cuda::make_device_ptr<float>(
         avgReference_.size());
   auto mask_d = glados::cuda::make_device_ptr<float>(
         numberOfDetectors_ * numberOfProjections_);
   CHECK(
         hipMemcpyAsync(avgDark_d.get(), avgDark_.data(),
               sizeof(float) * avgDark_.size(), hipMemcpyHostToDevice,
               streams_[deviceID]));
   CHECK(
         hipMemcpyAsync(avgReference_d.get(), avgReference_.data(),
               sizeof(float) * avgReference_.size(), hipMemcpyHostToDevice,
               streams_[deviceID]));
   //compute mask for relevant area
   std::vector<float> mask;
   relevantAreaMask(mask);
   CHECK(
         hipMemcpyAsync(mask_d.get(), mask.data(), sizeof(float) * mask.size(),
               hipMemcpyHostToDevice, streams_[deviceID]));

   dim3 blocks(blockSize2D_, blockSize2D_);
   dim3 grids(std::ceil(numberOfDetectors_ / (float)blockSize2D_),
         std::ceil(numberOfProjections_ / (float)blockSize2D_));
   float temp = pow(10, -5);
   CHECK(hipStreamSynchronize(streams_[deviceID]));
   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::Attenuation: Running Thread for Device " << deviceID;

   while (true) {
      auto sinogram = sinograms_[deviceID].take();
      if (!sinogram.valid())
         break;
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Attenuation: Attenuationing image with Index " << sinogram.index();

      auto sino =
            glados::MemoryPool<deviceManagerType>::instance()->requestMemory(
                  memoryPoolIdxs_[deviceID]);

      computeAttenuation<<<grids, blocks, 0, streams_[deviceID]>>>(
            sinogram.container().get(), mask_d.get(), sino.container().get(),
            avgReference_d.get(), avgDark_d.get(), temp, numberOfDetectors_,
            numberOfProjections_, sinogram.plane());
      CHECK(hipPeekAtLastError());

      sino.setIdx(sinogram.index());
      sino.setDevice(deviceID);
      sino.setPlane(sinogram.plane());
      sino.setStart(sinogram.start());

      //wait until work on device is finished
      CHECK(hipStreamSynchronize(streams_[deviceID]));
      results_.push(std::move(sino));

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Attenuation: Attenuationing image with Index " << sinogram.index() << " finished.";
   }
}

auto Attenuation::init() -> void {
   //create filter function
   std::vector<double> filterFunction{0.5, 1.0, 1.0, 1.0, 1.5, 2.0, 3.0, 3.5, 2.0, 3.5, 3.0, 2.0, 1.5, 1.0, 1.0, 1.0, 0.5};
   double sum = std::accumulate(filterFunction.cbegin(), filterFunction.cend(), 0.0);
   std::transform(filterFunction.begin(), filterFunction.end(), filterFunction.begin(),
         std::bind1st(std::multiplies<double>(), 1.0/sum));

   //read and average reference input values
   std::vector<unsigned short> referenceValues;
   if(pathReference_.back() != '/')
      pathReference_.append("/");
   std::string refPath = pathReference_ + "ref_empty_tomograph_repaired_DetModNr_";
   readInput(refPath, referenceValues, numberOfRefFrames_);
   //interpolate reference measurement
   for(auto i = 0; i < numberOfRefFrames_*numberOfPlanes_; i++){
      std::vector<int> defectDetectors(numberOfProjections_*numberOfDetectors_);
      findDefectDetectors(referenceValues.data()+i*numberOfDetectors_*numberOfProjections_, filterFunction, defectDetectors, numberOfDetectors_, numberOfProjections_,
         threshMin_, threshMax_);
      interpolateDefectDetectors(referenceValues.data()+i*numberOfDetectors_*numberOfProjections_, defectDetectors, numberOfDetectors_, numberOfProjections_);
   }
   computeAverage(referenceValues, avgReference_);

   //read and average dark input values
   std::vector<unsigned short> darkValues;
   if(pathDark_.back() != '/')
      pathDark_.append("/");
   std::string darkPath = pathDark_ + "dark_192.168.100_DetModNr_";
   readInput(darkPath, darkValues, numberOfDarkFrames_);
   computeDarkAverage(darkValues, avgDark_);
   //interpolate dark average
   for(auto j = 0; j < numberOfPlanes_; j++){
      for(auto i = 0; i < numberOfDetectors_; i++){
         if(avgDark_[i + j * numberOfDetectors_] > 300.0){
            BOOST_LOG_TRIVIAL(info) << "Interpolating dark value at detector " << i << " in plane " << j;
            avgDark_[numberOfDetectors_ * j + i] =
                                 0.5 * (avgDark_[numberOfDetectors_ * j + (i + 1)%numberOfDetectors_] +
                                       avgDark_[numberOfDetectors_ * j + (i - 1)%numberOfDetectors_]);
         }
      }
   }
}

template <typename T>
auto Attenuation::computeDarkAverage(const std::vector<T>& values, std::vector<float>& average) -> void {
   average.resize(numberOfDetectors_*numberOfPlanes_, 0.0);
   float factor = 1.0/ (float)((float)numberOfDarkFrames_*(float)numberOfProjections_);
   factor = 0.0;
   for(auto i = 0; i < numberOfDarkFrames_; i++){
      for(auto planeInd = 0; planeInd < numberOfPlanes_; planeInd++){
         for(auto detInd = 0; detInd < numberOfDetectors_; detInd++){
            for(auto projInd = 0; projInd < numberOfProjections_; projInd++){
               const float val = (float)values[detInd + numberOfDetectors_*projInd + (i*numberOfPlanes_+planeInd)*numberOfDetectors_*numberOfProjections_];
               average[detInd + planeInd*numberOfDetectors_] += val * factor;
            }
         }
      }
   }
}

template<typename T>
auto Attenuation::computeAverage(const std::vector<T>& values,
      std::vector<float>&average) -> void {
   average.resize(numberOfProjections_ * numberOfDetectors_ * numberOfPlanes_);
   float factor = 1.0 / (float) numberOfRefFrames_;
   for (auto i = 0; i < numberOfRefFrames_; i++) {
      for (auto planeInd = 0; planeInd < numberOfPlanes_; planeInd++) {
         for (auto index = 0; index < numberOfDetectors_ * numberOfProjections_;
               index++) {
            average[index + planeInd * numberOfDetectors_ * numberOfProjections_] +=
                  values[(i + planeInd) * numberOfProjections_
                        * numberOfDetectors_ + index] * factor;
         }
      }
   }
}

template<typename T>
auto Attenuation::readDarkInputFiles(std::string& path,
      std::vector<T>& values) -> void {
   //if(path.back() != '/')
   //   path.append("/");
   std::ifstream input(path + "dark_192.168.100.fxc",
         std::ios::in | std::ios::binary);
   if (!input) {
      BOOST_LOG_TRIVIAL(error)<< "recoLib::cuda::Attenuation: Source file could not be loaded.";
      throw std::runtime_error("File could not be opened. Please check!");
   }
   //allocate memory in vector
   std::streampos fileSize;
   input.seekg(0, std::ios::end);
   fileSize = input.tellg();
   input.seekg(0, std::ios::beg);
   values.resize(numberOfDetectors_ * numberOfPlanes_);
   input.read((char*) &values[0],
         numberOfDetectors_ * numberOfPlanes_ * sizeof(T));
}

template<typename T>
auto Attenuation::readInput(std::string& path,
      std::vector<T>& values, const int numberOfFrames) -> void {
   std::vector<std::vector<T>> fileContents(numberOfDetectorModules_);
   Timer tmr1, tmr2;
   //if(path.back() != '/')
   //   path.append("/");
   tmr1.start();
   tmr2.start();
#pragma omp parallel for default(shared) //num_threads(9)
   for (auto i = 1; i <= numberOfDetectorModules_; i++) {
      std::vector<T> content;
      //TODO: make filename and ending configurable
      std::ifstream input(path + std::to_string(i) + ".fx", std::ios::in | std::ios::binary);
      if (!input) {
         BOOST_LOG_TRIVIAL(error)<< "recoLib::cuda::Attenuation: Source file " << path + std::to_string(i) + ".fx" << " could not be loaded.";
         throw std::runtime_error("File could not be opened. Please check!");
      }
      //allocate memory in vector
      std::streampos fileSize;
      input.seekg(0, std::ios::end);
      fileSize = input.tellg();
      input.seekg(0, std::ios::beg);
      content.resize(fileSize / sizeof(T));
      input.read((char*) &content[0], fileSize);
      fileContents[i - 1] = content;
   }
   tmr2.stop();
   int numberOfDetPerModule = numberOfDetectors_ / numberOfDetectorModules_;
   values.resize(fileContents[0].size() * numberOfDetectorModules_);
   for (auto i = 0; i < numberOfFrames; i++) {
      for (auto planeInd = 0; planeInd < numberOfPlanes_; planeInd++) {
         for (auto projInd = 0; projInd < numberOfProjections_; projInd++) {
            for (auto detModInd = 0; detModInd < numberOfDetectorModules_;
                  detModInd++) {
               unsigned int startIndex = projInd * numberOfDetPerModule
                     + (planeInd + i * numberOfPlanes_) * numberOfDetPerModule * numberOfProjections_;
               unsigned int indexSorted = detModInd * numberOfDetPerModule
                     + projInd * numberOfDetectors_
                     + (planeInd + i * numberOfPlanes_) * numberOfDetectors_ * numberOfProjections_;
               std::copy(fileContents[detModInd].begin() + startIndex,
                     fileContents[detModInd].begin() + startIndex
                           + numberOfDetPerModule,
                     values.begin() + indexSorted);
            }
         }
      }
   }
   tmr1.stop();
   double totalFileSize = numberOfProjections_*numberOfDetectors_*numberOfPlanes_*numberOfRefFrames_*sizeof(unsigned short)/1024.0/1024.0;
   BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::Attenuation: Reading and sorting reference input took " << tmr1.elapsed() << " s, " << totalFileSize/tmr2.elapsed() << " MByte/s.";
}

template<typename T>
auto Attenuation::relevantAreaMask(std::vector<T>& mask) -> void {
   unsigned int ya, yb, yc, yd, ye;
   unsigned int yMin, yMax;
   double lowerLimit = (lowerLimOffset_ + sourceOffset_) / 360.0;
   double upperLimit = (upperLimOffset_ + sourceOffset_) / 360.0;
   //fill whole mask with ones and mask out the unrelevant parts afterwards
   mask.resize(numberOfProjections_ * numberOfDetectors_);
   std::fill(mask.begin(), mask.end(), 1.0);

   ya = std::round(lowerLimit * numberOfProjections_);
   yb = ya;
   yc = std::round(upperLimit * numberOfProjections_);
   yd = yc;

   //slope of the straight
   double m = ((double)ya - (double)yd) / ((double)xa_ - (double)xd_);

   ye = std::round((double)yc + ((double)xe_ - (double)xc_) * m);

   for (unsigned int x = 0; x <= xa_; x++) {
      yMin = ya;
      yMax = std::round(ye + m * x);
      for (auto y = yMin; y < yMax; y++)
         mask[x + y * numberOfDetectors_] = 0.0;
   }

   for (auto x = xa_; x <= xc_; x++) {
      yMin = std::round(ya + m * (x - xa_));
      yMax = std::round(ye + m * x);
      for (auto y = yMin; y < yMax; y++)
         mask[x + y * numberOfDetectors_] = 0.0;
   }

   for (auto x = xc_; x <= xd_; x++) {
      yMin = std::round(ya + m * (x - xa_));
      yMax = yd;
      for (auto y = yMin; y < yMax; y++)
         mask[x + y * numberOfDetectors_] = 0.0;
   }

   for (auto x = xb_; x <= xf_; x++) {
      yMin = yb;
      yMax = std::round(yb + m * (x - xb_));
      for (auto y = yMin; y < yMax; y++)
         mask[x + y * numberOfDetectors_] = 0.0;
   }

   std::fill(mask.begin(),
         mask.begin() + lowerLimit * numberOfDetectors_ * numberOfProjections_,
         0.0);
   std::fill(
         mask.begin() + upperLimit * numberOfProjections_ * numberOfDetectors_,
         mask.end(), 0.0);
}

auto Attenuation::readConfig(const read_json& config_reader) -> bool {
	int sampling_rate, scan_rate;
	try {
		numberOfDetectors_ = config_reader.get_value<int>("number_of_fan_detectors");
		numberOfDetectorModules_ = config_reader.get_value<int>("number_of_det_modules");
		numberOfRefFrames_ = config_reader.get_value<int>("number_of_reference_frames");
		pathDark_ = config_reader.get_element_in_list<std::string, std::string>("inputs", "inputpath", std::make_pair("inputtype", "dark"));
		pathReference_ = config_reader.get_element_in_list<std::string, std::string>("inputs", "inputpath", std::make_pair("inputtype", "reference"));
		numberOfPlanes_ = config_reader.get_value<int>("number_of_planes");
		sampling_rate = config_reader.get_value<int>("sampling_rate");
		scan_rate = config_reader.get_value<int>("scan_rate");
		sourceOffset_ = config_reader.get_value<float>("source_offset");
		xa_ = config_reader.get_value<unsigned int>("xa");
		xb_ = config_reader.get_value<unsigned int>("xb");
		xc_ = config_reader.get_value<unsigned int>("xc");
		xd_ = config_reader.get_value<unsigned int>("xd");
		xe_ = config_reader.get_value<unsigned int>("xe");
		xf_ = config_reader.get_value<unsigned int>("xf");
		lowerLimOffset_ = config_reader.get_value<double>("lower_lim_offset");
		upperLimOffset_ = config_reader.get_value<double>("upper_lim_offset");
		blockSize2D_ = config_reader.get_value<int>("blocksize_2d_attenutation");
		memPoolSize_ = config_reader.get_value<int>("mempoolsize_attenuation");
		threshMin_ = config_reader.get_value<double>("thresh_min");
		threshMax_ = config_reader.get_value<double>("thresh_max");
	} catch (const boost::property_tree::ptree_error& e) {
		BOOST_LOG_TRIVIAL(error) << "risa::cuda:Attenuation: Failed to read config: " << e.what();
		return EXIT_FAILURE;
	}
	numberOfProjections_ = sampling_rate * 1000000 / scan_rate;
	return EXIT_SUCCESS;
}

__global__ void computeAttenuation(
      const unsigned short* __restrict__ sinogram_in,
      const float* __restrict__ mask, float* __restrict__ sinogram_out,
      const float* __restrict__ avgReference, const float* __restrict__ avgDark,
      const float temp, const int numberOfDetectors,
      const int numberOfProjections, const int planeId) {

   auto x = glados::cuda::getX();
   auto y = glados::cuda::getY();
   if (x >= numberOfDetectors || y >= numberOfProjections)
      return;

   auto sinoIndex = numberOfDetectors * y + x;

   float numerator = (float) (sinogram_in[sinoIndex])
         - avgDark[planeId * numberOfDetectors + x];

   float denominator = avgReference[planeId * numberOfDetectors * numberOfProjections + sinoIndex]
         - avgDark[planeId * numberOfDetectors + x];

   if (numerator < temp)
      numerator = temp;
   if (denominator < temp)
      denominator = temp;

   //comutes the attenuation and multiplies with mask for hiding the unrelevant region
   sinogram_out[sinoIndex] = -log(numerator / denominator) * mask[sinoIndex];

}

}
}
