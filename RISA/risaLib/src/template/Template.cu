#include "hip/hip_runtime.h"
/*
 * This file is part of the RISA-library.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * RISA is free software: You can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * RISA is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with RISA. If not, see <http://www.gnu.org/licenses/>.
 *
 * Date: 30 November 2016
 * Authors: Tobias Frust (FWCC) <t.frust@hzdr.de>
 *
 */

#include "../../include/risa/template/Template.h"

#include <glados/cuda/Check.h>
#include <glados/MemoryPool.h>

#include <boost/log/trivial.hpp>

#include <nvToolsExt.h>

#include <pthread.h>
#include <exception>

namespace risa {
namespace cuda {

Template::Template(const std::string& configFile){

   read_json config_reader{};
   config_reader.read(configFile);
   if (!readConfig(config_reader)) {
      throw std::runtime_error(
            "recoLib::cuda::Template: unable to read config file. Please check!");
   }

   CHECK(hipGetDeviceCount(&numberOfDevices_));

   //when MemoryPool is required, register here:
   //memoryPoolIdx_ =
   //      glados::MemoryPool<hostManagerType>::instance()->registerStage(memPoolSize_,
   //            numberOfPixels_ * numberOfPixels_);

   //custom streams are necessary, because profiling with nvprof not possible with
   //-default-stream per-thread option
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      hipStream_t stream;
      CHECK(hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, 0));
      streams_[i] = stream;
   }

   //initialize worker threads
   for (auto i = 0; i < numberOfDevices_; i++) {
      processorThreads_[i] = std::thread { &Template::processor, this, i };
   }

   BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Template: Running " << numberOfDevices_ << " Threads.";
}

Template::~Template() {
   //when Memorypool was used, free memory here
   //glados::MemoryPool<hostManagerType>::instance()->freeMemory(memoryPoolIdx_);
   //when use of cudaStreams, destroy them here
   //for(auto i = 0; i < numberOfDevices_; i++){
   //   CHECK(hipSetDevice(i));
   //   CHECK(hipStreamDestroy(streams_[i]));
   //}
}

auto Template::process(input_type&& img) -> void {
   if (img.valid()) {
      BOOST_LOG_TRIVIAL(debug)<< "risa::cuda::Template: Image " << img.index() << "from device " << img.device() << "arrived";
      imgs_[img.device()].push(std::move(img));
   } else {
      BOOST_LOG_TRIVIAL(debug)<< "risa::cuda::Template: Received sentinel, finishing.";

      //send sentinal to processor threads and wait 'til they're finished
      for(auto i = 0; i < numberOfDevices_; i++) {
         imgs_[i].push(input_type());
      }

      for(auto i = 0; i < numberOfDevices_; i++) {
         processorThreads_[i].join();
      }

      //push sentinel to results for next stage
      results_.push(output_type());
      BOOST_LOG_TRIVIAL(info) << "risa::cuda::Template: Finished.";
   }
}

auto Template::wait() -> output_type {
   return results_.take();
}

auto Template::processor(const int deviceID) -> void {
   CHECK(hipSetDevice(deviceID));
   BOOST_LOG_TRIVIAL(info) << "recoLib::cuda::Template: Running Thread for Device " << deviceID;
   while (true) {
      auto img = imgs_[deviceID].take();
      if (!img.valid()) {
         BOOST_LOG_TRIVIAL(debug)<< "invalid image arrived.";
         break;
      }

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Template: ";

      //if necessary, request memory from MemoryPool here
      auto ret = glados::MemoryPool<hostManagerType>::instance()->requestMemory(
            memoryPoolIdx_);

      //<-- do work here -->

      //in case of a CUDA stage, synchronization needs to be done here
      //CHECK(hipStreamSynchronize(streams_[deviceID]));

      //wait until work on device is finished
      results_.push(std::move(ret));

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::Template: ";
   }
}

auto Template::readConfig(const read_json& config_reader) -> bool {
	try {
		numberOfPixels_ = config_reader.get_value<int>("number_of_pixels");
	} catch (const boost::property_tree::ptree_error& e) {
	   BOOST_LOG_TRIVIAL(error) << "risa::cuda::Template: Failed to read config: " << e.what();
	   return EXIT_FAILURE;
	}
	return EXIT_SUCCESS;
}

}
}
