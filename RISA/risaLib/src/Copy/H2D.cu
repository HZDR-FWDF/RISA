#include "hip/hip_runtime.h"
/*
 * This file is part of the RISA-library.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * RISA is free software: You can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * RISA is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with RISA. If not, see <http://www.gnu.org/licenses/>.
 *
 * Date: 30 November 2016
 * Authors: Tobias Frust (FWCC) <t.frust@hzdr.de>
 *
 */

#include "../../include/risa/Copy/H2D.h"

#include <glados/cuda/Coordinates.h>
#include <glados/cuda/Check.h>
#include <glados/MemoryPool.h>

#include <boost/log/trivial.hpp>

#include <nvToolsExt.h>

#include <exception>
#include <pthread.h>

namespace risa {
namespace cuda {

H2D::H2D(const std::string& config_file) : lastDevice_{0}, worstCaseTime_{0.0}, bestCaseTime_{std::numeric_limits<double>::max()},
      lastIndex_{0u}, lostSinos_{0u}{

   risa::read_json config_reader{};
   config_reader.read(config_file);
   if (readConfig(config_reader)) {
      throw std::runtime_error(
            "Configuration file could not be read. Please check!");
   }
   CHECK(hipGetDeviceCount(&numberOfDevices_));

   //allocate memory on all available devices
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      memoryPoolIdxs_[i] =
            glados::MemoryPool<deviceManagerType>::instance()->registerStage(memPoolSize_,
                  numberOfDetectors_ * numberOfProjections_);
      //custom streams are necessary, because profiling with nvprof seems to be
      //not possible with -default-stream per-thread option
      hipStream_t stream;
      CHECK(hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, 7));
      streams_[i] = stream;
   }

   //initialize worker threads
   for (auto i = 0; i < numberOfDevices_; i++) {
      processorThreads_[i] = std::thread { &H2D::processor, this, i };
   }

   BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::H2D: Running " << numberOfDevices_ << " Threads.";
}

H2D::~H2D() {
   for (auto idx : memoryPoolIdxs_) {
      CHECK(hipSetDevice(idx.first));
      glados::MemoryPool<deviceManagerType>::instance()->freeMemory(idx.second);
   }
   for(auto i = 0; i < numberOfDevices_; i++){
      CHECK(hipSetDevice(i));
      CHECK(hipStreamDestroy(streams_[i]));
   }
   BOOST_LOG_TRIVIAL(info) << "WorstCaseTime: " << worstCaseTime_ << "s; BestCaseTime: " << bestCaseTime_ << "s;";
   BOOST_LOG_TRIVIAL(info) << "Could not reconstruct " << lostSinos_ << " elements; " << lostSinos_/(double)lastIndex_*100.0 << "% loss";
}

auto H2D::process(input_type&& sinogram) -> void {
   if (sinogram.valid()) {
      if(sinogram.index() > 0)
         tmr_.stop();
      BOOST_LOG_TRIVIAL(debug) << "H2D: Image arrived with Index: " << sinogram.index() << "to device " << lastDevice_;
//      int device = sinogram.index() % 5;
//      if(device == 0) device = 1;
//      else device = 0;
      sinograms_[lastDevice_].push(std::move(sinogram));
      lastDevice_ = (lastDevice_ + 1) % numberOfDevices_;
      double time = tmr_.elapsed();
      if(sinogram.index() > 0){
         if(time < bestCaseTime_)
            bestCaseTime_ = time;
         if(time > worstCaseTime_)
            worstCaseTime_ = time;
      }
      tmr_.start();
      int diff = sinogram.index() - lastIndex_ - 1;
      lostSinos_ += diff;
      if(diff > 0)
         BOOST_LOG_TRIVIAL(debug) << "Skipping " << diff << " elements.";
      if(count_%10000 == 0)
         BOOST_LOG_TRIVIAL(info) << "Did not process " << lostSinos_ << " elements; " << lostSinos_/(double)lastIndex_*100.0 << "% loss";
      count_++;
      lastIndex_ = sinogram.index();
   } else {
      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::H2D: Received sentinel, finishing.";

      //send sentinel to all processor threads and wait 'til they're finished
      for(auto i = 0; i < numberOfDevices_; i++){
         sinograms_[i].push(input_type());
      }

      //wait until all threads are finished
      for(auto i = 0; i < numberOfDevices_; i++){
         processorThreads_[i].join();
      }

      //push sentinel to results for next stage
      results_.push(output_type());

      BOOST_LOG_TRIVIAL(info)<< "recoLib::cuda::H2D: Finished.";
   }
}

auto H2D::wait() -> output_type {
   return results_.take();
}

auto H2D::processor(int deviceID) -> void {
   //nvtxNameOsThreadA(pthread_self(), "H2D");
   CHECK(hipSetDevice(deviceID));
   //for conversion from short to float
   std::vector<float> temp(numberOfProjections_*numberOfDetectors_);
   auto inputShort_d = glados::cuda::make_device_ptr<unsigned short>(numberOfProjections_*numberOfDetectors_);
   BOOST_LOG_TRIVIAL(info) << "recoLib::cuda::H2D: Running Thread for Device " << deviceID;
   while (true) {
      auto sinogram = sinograms_[deviceID].take();
      if (!sinogram.valid())
         break;

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::H2D: Copy sinogram " << sinogram.index() << " to device " << deviceID;

      //copy image from device to host
      auto img = glados::MemoryPool<deviceManagerType>::instance()->requestMemory(
            memoryPoolIdxs_[deviceID]);

      CHECK(
            hipMemcpyAsync(img.container().get(),sinogram.container().get(),
                   sinogram.size() * sizeof(unsigned short), hipMemcpyHostToDevice, streams_[deviceID]));

      //needs to be set due to reuse of memory
      img.setIdx(sinogram.index());
      img.setDevice(deviceID);
      img.setPlane(sinogram.plane());
      img.setStart(sinogram.start());

      CHECK(hipStreamSynchronize(streams_[deviceID]));

      //wait until work on device is finished
      results_.push(std::move(img));

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::H2D: Copy sinogram " << sinogram.index() << " to device finished.";
   }
}

auto H2D::readConfig(const read_json& config_reader) -> bool {
   int sampling_rate, scan_rate;
   try {
	   numberOfDetectors_ = config_reader.get_value<int>("number_of_fan_detectors");
	   memPoolSize_ = config_reader.get_value<int>("mempoolsize_h2d");
	   sampling_rate = config_reader.get_value<int>("sampling_rate");
	   scan_rate = config_reader.get_value<int>("scan_rate");
   } catch (const boost::property_tree::ptree_error& e) {
	   BOOST_LOG_TRIVIAL(error) << "risa::cuda:H2D: Failed to read config: " << e.what();
	   return EXIT_FAILURE;
   }
   numberOfProjections_ = sampling_rate * 1000000 / scan_rate;
   return EXIT_SUCCESS;

}

}
}
