#include "hip/hip_runtime.h"
/*
 * This file is part of the RISA-library.
 *
 * Copyright (C) 2016 Helmholtz-Zentrum Dresden-Rossendorf
 *
 * RISA is free software: You can redistribute it and/or modify
 * it under the terms of the GNU Lesser General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * RISA is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
 * GNU Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public License
 * along with RISA. If not, see <http://www.gnu.org/licenses/>.
 *
 * Date: 30 November 2016
 * Authors: Tobias Frust <t.frust@hzdr.de>
 *
 */

#include "../../include/risa/Copy/D2H.h"

#include <glados/cuda/Check.h>
#include <glados/MemoryPool.h>

#include <boost/log/trivial.hpp>

#include <nvToolsExt.h>

#include <pthread.h>
#include <exception>

namespace risa {
namespace cuda {

D2H::D2H(const std::string& config_file) : reconstructionRate_(0), counter_(1.0){

   risa::read_json config_reader{};
   config_reader.read(config_file);
   if (readConfig(config_reader)) {
      throw std::runtime_error(
            "recoLib::cuda::D2H: unable to read config file. Please check!");
   }

   CHECK(hipGetDeviceCount(&numberOfDevices_));

   memoryPoolIdx_ =
         glados::MemoryPool<hostManagerType>::instance()->registerStage(memPoolSize_,
               numberOfPixels_ * numberOfPixels_);

//   memoryPoolIdx_ =
//        glados::MemoryPool<hostManagerType>::instance()->registerStage(memPoolSize_,
//               256*1024);

   //custom streams are necessary, because profiling with nvprof not possible with
   //-default-stream per-thread option
   for (auto i = 0; i < numberOfDevices_; i++) {
      CHECK(hipSetDevice(i));
      hipStream_t stream;
      CHECK(hipStreamCreateWithPriority(&stream, hipStreamNonBlocking, 0));
      streams_[i] = stream;
   }

   //initialize worker threads
   for (auto i = 0; i < numberOfDevices_; i++) {
      processorThreads_[i] = std::thread { &D2H::processor, this, i };
   }
   BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::D2H: Running " << numberOfDevices_ << " Threads.";
   tmr_.start();
}

D2H::~D2H() {
   BOOST_LOG_TRIVIAL(info) << "Reconstructed " << reconstructionRate_ << " Images/s in average.";
   glados::MemoryPool<hostManagerType>::instance()->freeMemory(memoryPoolIdx_);
   for(auto i = 0; i < numberOfDevices_; i++){
      CHECK(hipSetDevice(i));
      CHECK(hipStreamDestroy(streams_[i]));
   }
}

auto D2H::process(input_type&& img) -> void {
   if (img.valid()) {
      if(img.index() == 0)
         tmr_.start();
      if((count_ % 10000) == 9999){
         tmr_.stop();
         reconstructionRate_ = (reconstructionRate_*(counter_-1.0) + 10000.0/(tmr_.elapsed())) / counter_;
         counter_ += 1.0;
         BOOST_LOG_TRIVIAL(info) << "Reconstructing at " << 10000.0/(tmr_.elapsed()) << " Images/second.";
         tmr_.start();
      }
      count_++;
      BOOST_LOG_TRIVIAL(debug)<< "Image " << img.index() << "from device " << img.device() << "arrived";
      imgs_[img.device()].push(std::move(img));
   } else {
      BOOST_LOG_TRIVIAL(debug)<< "cuda::D2H: Received sentinel, finishing.";

      //send sentinal to processor threads and wait 'til they're finished
      for(auto i = 0; i < numberOfDevices_; i++) {
         imgs_[i].push(input_type());
      }

      for(auto i = 0; i < numberOfDevices_; i++) {
         processorThreads_[i].join();
      }

      //push sentinel to results for next stage
      results_.push(output_type());
      BOOST_LOG_TRIVIAL(info) << "cuda::D2H: Finished.";
   }
}

auto D2H::wait() -> output_type {
   return results_.take();
}

auto D2H::processor(const int deviceID) -> void {
   //nvtxNameOsThreadA(pthread_self(), "D2H");
   CHECK(hipSetDevice(deviceID));
   BOOST_LOG_TRIVIAL(info) << "recoLib::cuda::D2H: Running Thread for Device " << deviceID;
   while (true) {
      auto img = imgs_[deviceID].take();
      if (!img.valid()) {
         BOOST_LOG_TRIVIAL(debug)<< "invalid image arrived.";
         break;
      }

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::D2H: Copy sinogram " << img.index() << " from device " << img.device();

      //copy image from device to host
      auto ret = glados::MemoryPool<hostManagerType>::instance()->requestMemory(
            memoryPoolIdx_);
      CHECK(
            hipMemcpyAsync(ret.container().get(), img.container().get(),
                  img.size() * sizeof(float), hipMemcpyDeviceToHost, streams_[deviceID]));
      ret.setIdx(img.index());
      ret.setPlane(img.plane());
      ret.setStart(img.start());
      CHECK(hipStreamSynchronize(streams_[deviceID]));

      //wait until work on device is finished
      results_.push(std::move(ret));

      BOOST_LOG_TRIVIAL(debug)<< "recoLib::cuda::D2H: Copy sinogram " << img.index() << " from device " << img.device() << " finished.";
   }
}

auto D2H::readConfig(const read_json& config_reader) -> bool {
   try {
	   numberOfPixels_ = config_reader.get_value<int>("number_of_pixels");
	   memPoolSize_ = config_reader.get_value<int>("mempoolsize_d2h");
   } catch (const boost::property_tree::ptree_error& e) {
	   BOOST_LOG_TRIVIAL(error) << "risa::cuda::D2H: Failed to read config: " << e.what();
	   return EXIT_FAILURE;
   }
   return EXIT_SUCCESS;
}

}
}
